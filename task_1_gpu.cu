#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>
#include "public/CheckMatrix.h"
#include "public/Utils.h"

// CUDA 内核函数：检查空行和单元素行
__global__ void CheckRowsKernel(double *B, int m, int n, RowInfo *rows, int *rowCount) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= m) return;

    bool isEmpty = true;
    bool isSingleton = false;
    int nonZeroCount = 0;
    int nonZeroCol = -1;

    for (int j = 0; j < n; ++j) {
        if (B[row * n + j] != 0) {
            isEmpty = false;
            nonZeroCount++;
            nonZeroCol = j;
        }
    }

    if (isEmpty || nonZeroCount == 1) {
        int idx = atomicAdd(rowCount, 1);
        rows[idx].type = isEmpty ? 0 : 1;
        rows[idx].irow = row;
        rows[idx].jcol = nonZeroCol;
    }
}

// CUDA 内核函数：检查空列和单元素列
__global__ void CheckColsKernel(double *C, int m, int n, ColInfo *cols, int *colCount) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= n) return;

    bool isEmpty = true;
    bool isSingleton = false;
    int nonZeroCount = 0;
    int nonZeroRow = -1;

    for (int i = 0; i < m; ++i) {
        if (C[i * n + col] != 0) {
            isEmpty = false;
            nonZeroCount++;
            nonZeroRow = i;
        }
    }

    if (isEmpty || nonZeroCount == 1) {
        int idx = atomicAdd(colCount, 1);
        cols[idx].type = isEmpty ? 0 : 1;
        cols[idx].jcol = col;
        cols[idx].irow = nonZeroRow;
    }
}

// 检查空行和单元素行
int CheckEmptyAndSingletonRows(double *B, int m, int n, RowInfo **Rows, int *RowsSize) {
    double *d_B;
    RowInfo *d_rows;
    int *d_rowCount;
    int rowCount = 0;

    hipMalloc((void**)&d_B, m * n * sizeof(double));
    hipMalloc((void**)&d_rows, m * sizeof(RowInfo));
    hipMalloc((void**)&d_rowCount, sizeof(int));

    hipMemcpy(d_B, B, m * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_rowCount, &rowCount, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (m + blockSize - 1) / blockSize;

    CheckRowsKernel<<<numBlocks, blockSize>>>(d_B, m, n, d_rows, d_rowCount);

    hipMemcpy(&rowCount, d_rowCount, sizeof(int), hipMemcpyDeviceToHost);
    RowInfo *h_rows = (RowInfo*)malloc(rowCount * sizeof(RowInfo));
    hipMemcpy(h_rows, d_rows, rowCount * sizeof(RowInfo), hipMemcpyDeviceToHost);

    *Rows = h_rows;
    *RowsSize = rowCount;

    hipFree(d_B);
    hipFree(d_rows);
    hipFree(d_rowCount);

    return rowCount;
}

// 检查空列和单元素列
int CheckEmptyAndSingletonCols(double *C, int m, int n, ColInfo **Cols, int *ColsSize) {
    double *d_C;
    ColInfo *d_cols;
    int *d_colCount;
    int colCount = 0;

    hipMalloc((void**)&d_C, m * n * sizeof(double));
    hipMalloc((void**)&d_cols, n * sizeof(ColInfo));
    hipMalloc((void**)&d_colCount, sizeof(int));

    hipMemcpy(d_C, C, m * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_colCount, &colCount, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    CheckColsKernel<<<numBlocks, blockSize>>>(d_C, m, n, d_cols, d_colCount);

    hipMemcpy(&colCount, d_colCount, sizeof(int), hipMemcpyDeviceToHost);
    ColInfo *h_cols = (ColInfo*)malloc(colCount * sizeof(ColInfo));
    hipMemcpy(h_cols, d_cols, colCount * sizeof(ColInfo), hipMemcpyDeviceToHost);

    *Cols = h_cols;
    *ColsSize = colCount;

    hipFree(d_C);
    hipFree(d_cols);
    hipFree(d_colCount);

    return colCount;
}


int main(int argc, char *argv[]) {
    char *filename = "../A(2262x9799).80bau3b.bin";
    int M = 2262;
    int N = 9799;
    if (argc == 4) {
        filename = argv[1];
        M = atoi(argv[2]);
        N = atoi(argv[3]);
    } else if (argc != 4 && argc != 1) {
        return 0;
    }

    int nThread = 8; // 线程数
    double total_time_used = 0.0;
    int iterations = 500; // 执行测试的次数

    // 动态分配内存
    double **pA = (double **)malloc(M * sizeof(double *));
    double *A = (double *)malloc(M * N * sizeof(double));
    double *B = (double *)malloc(M * sizeof(double));
    double *C = (double *)malloc(N * sizeof(double));
    RowInfo *Rows;
    ColInfo *Cols;
    int RowsSize = 0; // 用于存储空行和单元素行的数量
    int ColsSize = 0; // 用于存储空列和单元素列的数量
    for (int i = 0; i < M; i++) {
        pA[i] = &(A[i * N]);
    }
    // 从文件读取矩阵数据
    ReadMatrix(pA, M, N,filename);

    // 执行测试并计时
    for (int iter = 0; iter < iterations; ++iter) {
        auto start = std::chrono::high_resolution_clock::now();

        RowsSize = 0;
        ColsSize = 0;
        CheckEmptyAndSingletonRows(A, M, N, &Rows, &RowsSize);
        CheckEmptyAndSingletonCols(A, M, N, &Cols, &ColsSize);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        total_time_used += elapsed.count();
        // // 输出检测结果
        // if (iter == 0) { // 仅在第一次迭代输出检测结果
        //     std::cout << "Empty and singleton rows:\n";
        //     for (int i = 0; i < RowsSize; ++i) {
        //         std::cout << "Row " << Rows[i].irow << ", Type: " << (Rows[i].type == 0 ? "Empty" : "Singleton") << "\n";
        //     }
        //     std::cout << "Empty and singleton cols:\n";
        //     for (int i = 0; i < ColsSize; ++i) {
        //         std::cout << "Col " << Cols[i].jcol << ", Type: " << (Cols[i].type == 0 ? "Empty" : "Singleton") << "\n";
        //     }
        // }
    }

    double average_time_used = total_time_used / iterations;
    printf("M: %d, N: %d, GPU多线程平均函数执行耗时: %f 秒\n", M, N, average_time_used);

    // 释放内存
    free(A);
    free(B);
    free(C);
    free(Rows);
    free(Cols);

    return 0;
}
